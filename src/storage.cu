#include "hip/hip_runtime.h"
#include <sys/stat.h>

#include <iomanip> // setprecision
#include <sstream> // stringstream

#include "system.h"
#include "system_builder.h"

#include "system_structures.h"
#include "storage.h"
#include <numeric>

Storage::Storage(std::weak_ptr<System> a_system,
	std::weak_ptr<System_Builder> b_system , const std::string& a_fileName) {
	
	system = a_system;
	builder = b_system;
	bn = a_fileName; //this will be used later to open files
	//std::ofstream statesOutput(a_fileName + ".sta");
	//std::ofstream statesOutputStrain(a_fileName + "_Strain.sta");

	std::shared_ptr<System> sys = system.lock();

	if ( sys ){
		std::stringstream stream_min;
		std::stringstream stream_max;

		unsigned domain_size = ceil((sys->domainParams.max_x + 
			sys->domainParams.max_y + 
			sys->domainParams.max_z) / 3.0);

		unsigned max_nodes = sys->generalParams.max_node_count;
		unsigned max_z = sys->domainParams.max_z;
		unsigned max_x = sys->domainParams.max_x;
		unsigned axis = sys->extensionParams.axis;
		int pull_ammount = int(100*sys->generalParams.pull_ammount);
		//std::stringstream tmp;
		//tmp << std::setprecision(3) << std::fixed << pull_ammount;
		//pull_ammount = stod(tmp.str());
		
		int epsilon = int(1000*sys->generalParams.epsilon_factor);
		std::stringstream tmp1;
		tmp1 << std::setprecision(4) << std::fixed << epsilon;
		epsilon = stod(tmp1.str());

		std::string str_nodes = "_max_nodes_";
		std::string str_z = "_max_z_";
		std::string str_x = "_max_x_";
		std::string str_axis = "_axis_";
		std::string str_pull = "_pullwidth_";
		std::string str_eps = "_eps_";

		std::string str_a = "Animation_";
		std::string str_p = "Params_";
		
		str_animation = str_a +
			str_nodes + std::to_string(max_nodes)+
			str_z + std::to_string(max_z)+
			str_x + std::to_string(max_x)+
			str_axis + std::to_string(axis)+
			str_pull + std::to_string(pull_ammount)+
			str_eps + std::to_string(epsilon);

		const int dir_err_anim = mkdir(str_animation.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
		if (-1 == dir_err_anim)
		{printf("Error creating directory animation test!n");}
		else {printf("making folder!n"); printf(str_animation.c_str());}

		str_params = str_p + 			
			str_nodes + std::to_string(max_nodes)+
			str_z + std::to_string(max_z)+
			str_x + std::to_string(max_x)+
			str_axis + std::to_string(axis)+
			str_pull + std::to_string(pull_ammount)+
			str_eps + std::to_string(epsilon);

		const int dir_err_params = mkdir(str_params.c_str(), S_IRWXU | S_IRWXG | S_IROTH | S_IXOTH);
		if (-1 == dir_err_params)
		{printf("Error creating directory params!n");}
		else {printf("making folder!n"); printf(str_params.c_str());}
	}
};

void Storage::save_params(void) {
	std::shared_ptr<System> sys = system.lock();
	if (sys) {

		double currentStrain = (sys->extensionParams.averageUpperStrain - sys->extensionParams.averageLowerStrain) /
			 (sys->extensionParams.originAverageUpperStrain - sys->extensionParams.originAverageLowerStrain ) - 1.0;
		//first create a new file using the current network strain

		std::string format = ".sta";
		
		std::string strain =  std::to_string(sys->generalParams.currentTime);
		std::string initial = str_params+"/Param_";
		std::ofstream ofs;
		std::string Filename = initial + strain + format;
		ofs.open(Filename.c_str());


		unsigned max_nbr_count = sys->generalParams.max_nbr_count;
		unsigned max_node_count = sys->generalParams.max_node_count;
		unsigned originalNodeCount = sys->generalParams.origin_node_count;
		unsigned originalEdgeCount = sys->generalParams.origin_edge_count;
		unsigned edgeCountDiscretize = sys->generalParams.current_edge_count;
		//Now first place strain
		ofs << std::setprecision(5) <<std::fixed<< "time " << sys->generalParams.currentTime<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "network_strain " << currentStrain<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_x " << sys->domainParams.min_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_x " << sys->domainParams.max_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_y " << sys->domainParams.min_y<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_y " << sys->domainParams.max_y<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "min_z " << sys->domainParams.min_x<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "max_z " << sys->domainParams.max_x<<std::endl;

		ofs << std::setprecision(5) <<std::fixed<< "force_upper " << sys->extensionParams.applied_force_upper<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "force_lower " << sys->extensionParams.applied_force_lower<<std::endl;


		//ofs << std::setprecision(5) <<std::fixed<< "total_applied_force " << sys->extensionParams.totalAppliedForce<<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_node_count " << originalNodeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "node_count_discretize " << max_node_count <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "original_edge_count " << originalEdgeCount <<std::endl;
		ofs << std::setprecision(5) <<std::fixed<< "edge_count_discretize " << edgeCountDiscretize <<std::endl;

		//place nodes
		for (unsigned i = 0; i < sys->nodeInfoVecs.node_loc_x.size(); i++) {
			double x = sys->nodeInfoVecs.node_loc_x[i];
			double y = sys->nodeInfoVecs.node_loc_y[i];
			double z = sys->nodeInfoVecs.node_loc_z[i];
			ofs << std::setprecision(5) <<std::fixed<< "node " << x << " " << y << " " << z <<std::endl;

		}
		//place force node is experiencing
		for (unsigned i = 0; i < sys->nodeInfoVecs.node_loc_x.size(); i++) {
			ofs << std::setprecision(5) <<std::fixed<< "force_on_node " << sys->nodeInfoVecs.sum_forces_on_node[i]<<std::endl;

		}

		//place original edges
		for (unsigned edge = 0; edge < sys->generalParams.origin_edge_count; edge++) {
			unsigned idL = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idR = sys->nodeInfoVecs.host_edge_right[edge];
			ofs <<"original_edge_discretized " <<idL <<" "<< idR <<std::endl;

		}

		//place added edges
		for (unsigned edge = sys->generalParams.origin_edge_count; edge < sys->generalParams.current_edge_count; edge++) {
			unsigned idL = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idR = sys->nodeInfoVecs.host_edge_right[edge];
			ofs <<"added_edge " <<idL <<" "<< idR <<std::endl;

		}

		//original edge strain
		for (unsigned i = 0; i < sys->generalParams.origin_edge_count; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_strain[i];

			ofs << std::setprecision(5)<< std::fixed<<"original_edge_strain " << val <<std::endl;
		}

		//original edge alignment
		for (unsigned i = 0; i < sys->generalParams.origin_edge_count; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_alignment[i];
			ofs << std::setprecision(5)<< std::fixed<<"original_edge_alignment " << val <<std::endl;
		}

		//added edge strain
		for (unsigned i = sys->generalParams.origin_edge_count; i < sys->generalParams.current_edge_count; i++ ){
			double val = sys->nodeInfoVecs.discretized_edges_strain[i];
			ofs << std::setprecision(5)<< std::fixed<<"added_edge_strain " << val <<std::endl;
		}

		//added links per node.
		for (unsigned i = 0; i < sys->generalParams.max_node_count; i++ ){
			unsigned val = sys->edgeInfoVecs.current_node_edge_count_vec[i] -
				sys->edgeInfoVecs.num_origin_nbr_per_node_vec[i];
			ofs << std::setprecision(5)<< std::fixed<<"bind_sites_per_node " << val <<std::endl;
		}



	}
}


void Storage::print_VTK_file() {

	std::shared_ptr<System> sys = system.lock();
	if (sys) {

		unsigned max_node_count = sys->generalParams.max_node_count;
		unsigned max_nbr_count = sys->generalParams.max_nbr_count;
		unsigned num_collagen_edges = 0;
		unsigned num_elastin_edges=0;
		unsigned numEdges = sys->nodeInfoVecs.host_edge_left.size();
		for (unsigned edge = 0; edge < numEdges; edge++) {	
			unsigned idA = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.host_edge_right[edge];
			bool is_A_collagen = sys->nodeInfoVecs.node_is_collagen[idA];
			bool is_B_collagen = sys->nodeInfoVecs.node_is_collagen[idB];
			if (is_A_collagen && is_B_collagen){ num_collagen_edges+=1;}
			else{num_elastin_edges+=1;}
		}
		
		iteration+=1;
		unsigned digits = ceil(log10(iteration + 1));
		std::string format = ".vtk";
		std::string Number;
		std::string initial = str_animation + "/Collagen_Network_";
		std::ofstream ofs;
		if (digits == 1 || digits == 0) {
			Number = "0000" + std::to_string(iteration);
		}
		else if (digits == 2) {
			Number = "000" + std::to_string(iteration);
		}
		else if (digits == 3) {
			Number = "00" + std::to_string(iteration);
		}
		else if (digits == 4) {
			Number = "0" + std::to_string(iteration);
		}

		std::string Filename = initial + Number + format;

		ofs.open(Filename.c_str());

		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;


		ofs << "POINTS " << max_node_count << " float" << std::endl;
		for (unsigned i = 0; i< max_node_count; i++) {
			double xPos = sys->nodeInfoVecs.node_loc_x[i];
			double yPos = sys->nodeInfoVecs.node_loc_y[i];
			double zPos = sys->nodeInfoVecs.node_loc_z[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}
		//now plot particles
		unsigned numCells = num_collagen_edges;
		unsigned numNumsInCells = 3 * num_collagen_edges;

		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;

		for (unsigned edge = 0; edge < numEdges; edge++) {
			
			unsigned idA = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.host_edge_right[edge];
			bool is_A_collagen = sys->nodeInfoVecs.node_is_collagen[idA];
			bool is_B_collagen = sys->nodeInfoVecs.node_is_collagen[idB];
			if (is_A_collagen && is_B_collagen){
				ofs<< 2 << " " << idA << " " << idB << std::endl;
			}
		}

		ofs << "CELL_TYPES " << numCells << std::endl;
		for (unsigned i = 0; i<num_collagen_edges; i++) {
			ofs << 3 << std::endl;
		}

		
		ofs << "CELL_DATA " << numCells << std::endl;
		ofs << "SCALARS Fiber_Strain double " << std::endl;
		ofs << "LOOKUP_TABLE default "  << std::endl;
		for (unsigned edge = 0; edge < numEdges; edge++) {
			unsigned idA = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.host_edge_right[edge];
			bool is_A_collagen = sys->nodeInfoVecs.node_is_collagen[idA];
			bool is_B_collagen = sys->nodeInfoVecs.node_is_collagen[idB];
			if (is_A_collagen && is_B_collagen){
				unsigned begin = idA * max_nbr_count;
				unsigned end = begin + max_nbr_count;
				double L0;
				for (unsigned i = begin; i < end; i++) {
					unsigned idTemp = sys->edgeInfoVecs.global_neighbors[i];
					if (idTemp == idB){
						L0 = sys->edgeInfoVecs.global_length_zero[i];
					}
				}
				double xL = sys->nodeInfoVecs.node_loc_x[idA];
				double yL = sys->nodeInfoVecs.node_loc_y[idA];
				double zL = sys->nodeInfoVecs.node_loc_z[idA];
				double xR = sys->nodeInfoVecs.node_loc_x[idB];
				double yR = sys->nodeInfoVecs.node_loc_y[idB];
				double zR = sys->nodeInfoVecs.node_loc_z[idB];

				double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
				double strain = (L1 - L0) / L0;
				ofs << std::fixed << strain   << std::endl;
			}

		}

		ofs.close();


		//Now print elastin
		
		initial = str_animation + "/Elastin_Network_";
		Filename = initial + Number + format;

		ofs.open(Filename.c_str());

		ofs << "# vtk DataFile Version 3.0" << std::endl;
		ofs << "Point representing Sub_cellular elem model" << std::endl;
		ofs << "ASCII" << std::endl << std::endl;
		ofs << "DATASET UNSTRUCTURED_GRID" << std::endl;


		ofs << "POINTS " << max_node_count << " float" << std::endl;
		for (unsigned i = 0; i< max_node_count; i++) {
			double xPos = sys->nodeInfoVecs.node_loc_x[i];
			double yPos = sys->nodeInfoVecs.node_loc_y[i];
			double zPos = sys->nodeInfoVecs.node_loc_z[i];

			ofs << std::setprecision(5) <<std::fixed<< xPos << " " << yPos << " " << zPos << " " << '\n'<< std::fixed;
		}
		//now plot particles
		numCells = num_elastin_edges;
		numNumsInCells = 3 * num_elastin_edges;

		ofs << "CELLS " << numCells << " " << numNumsInCells << std::endl;

		for (unsigned edge = 0; edge < numEdges; edge++) {
			
			unsigned idA = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.host_edge_right[edge];
			bool is_A_collagen = sys->nodeInfoVecs.node_is_collagen[idA];
			bool is_B_collagen = sys->nodeInfoVecs.node_is_collagen[idB];
			if ((!is_A_collagen) || (!is_B_collagen)){
				ofs<< 2 << " " << idA << " " << idB << std::endl;
			}
		}

		ofs << "CELL_TYPES " << numCells << std::endl;
		for (unsigned i = 0; i<num_elastin_edges; i++) {
			ofs << 3 << std::endl;
		}

		
		ofs << "CELL_DATA " << numCells << std::endl;
		ofs << "SCALARS Fiber_Strain double " << std::endl;
		ofs << "LOOKUP_TABLE default "  << std::endl;
		for (unsigned edge = 0; edge < numEdges; edge++) {
			unsigned idA = sys->nodeInfoVecs.host_edge_left[edge];
			unsigned idB = sys->nodeInfoVecs.host_edge_right[edge];
			bool is_A_collagen = sys->nodeInfoVecs.node_is_collagen[idA];
			bool is_B_collagen = sys->nodeInfoVecs.node_is_collagen[idB];
			if ((!is_A_collagen) || (!is_B_collagen) ){
				unsigned begin = idA * max_nbr_count;
				unsigned end = begin + max_nbr_count;
				double L0;
				for (unsigned i = begin; i < end; i++) {
					unsigned idTemp = sys->edgeInfoVecs.global_neighbors[i];
					if (idTemp == idB){
						L0 = sys->edgeInfoVecs.global_length_zero[i];
					}
				}
				double xL = sys->nodeInfoVecs.node_loc_x[idA];
				double yL = sys->nodeInfoVecs.node_loc_y[idA];
				double zL = sys->nodeInfoVecs.node_loc_z[idA];
				double xR = sys->nodeInfoVecs.node_loc_x[idB];
				double yR = sys->nodeInfoVecs.node_loc_y[idB];
				double zR = sys->nodeInfoVecs.node_loc_z[idB];

				double L1 = std::sqrt( (xL - xR)*(xL - xR)+(yL - yR)*(yL - yR)+(zL - zR)*(zL - zR));
				double strain = (L1 - L0) / L0;
				ofs << std::fixed << strain   << std::endl;
			}

		}

		ofs.close();

	}
};
