#include "hip/hip_runtime.h"

#include "link_nodes.h"

#include "system_structures.h"
#include "system.h"

#include "functor_de_link_nodes.h"
#include "functor_link_nodes.h"


void link_nodes(
	NodeInfoVecs& nodeInfoVecs,
	EdgeInfoVecs& edgeInfoVecs,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

		//Default fill values at 0.
		thrust::fill(nodeInfoVecs.links_made_individual_thread.begin(),
			nodeInfoVecs.links_made_individual_thread.end(), 0);

		thrust::fill(nodeInfoVecs.id_temp_linked_left.begin(),
				nodeInfoVecs.id_temp_linked_left.end(), 0);

		thrust::fill(nodeInfoVecs.id_temp_linked_right.begin(),
				nodeInfoVecs.id_temp_linked_right.end(), 0);

		thrust::counting_iterator<unsigned> counter(0);

		thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket_net_intc.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						counter,
						auxVecs.id_bucket_net_intc.begin())) + generalParams.max_node_count,
				nodeInfoVecs.links_made_individual_thread.begin(),//output
			functor_link_nodes(
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_is_collagen.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.node_is_elastin.data()),

				thrust::raw_pointer_cast(edgeInfoVecs.current_node_edge_count_vec.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_neighbors.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_length_zero.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_isedge_collagen.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_isedge_elastin.data()),

				thrust::raw_pointer_cast(auxVecs.id_value_expanded_net_intc.data()),
				thrust::raw_pointer_cast(auxVecs.key_begin_net_intc.data()),
				thrust::raw_pointer_cast(auxVecs.key_end_net_intc.data()),

				edgeInfoVecs.collagen_diameter,
				edgeInfoVecs.elastin_diameter,
				generalParams.max_nbr_count,
				generalParams.max_node_count,

				generalParams.max_links_per_iteration,
				thrust::raw_pointer_cast(nodeInfoVecs.id_temp_linked_left.data()),
				thrust::raw_pointer_cast(nodeInfoVecs.id_temp_linked_right.data()) ) );


			/*
			for (unsigned i = 0; i < nodeInfoVecs.id_temp_linked_left.size(); i++) {
				unsigned varL = nodeInfoVecs.id_temp_linked_left[i];
				unsigned varR = nodeInfoVecs.id_temp_linked_right[i];

				if ((varL != 0) || (varR != 0))
					std::cout<< varL << " " <<varR << std::endl;
			}
			for (unsigned i = 0; i < edgeInfoVecs.global_neighbors.size(); i++) {
				if ((i > 0) && (i % ( generalParams.max_nbr_count) == 0)){
					std::cout << " " << std::endl;
				}
				unsigned varL = edgeInfoVecs.global_neighbors[i];
				if (varL < generalParams.max_node_count){
					std::cout << " " << varL;
				}
			}
			std::cout << " " << std::endl;
			std::cout<< nodeInfoVecs.node_loc_x[0] << " " << nodeInfoVecs.node_loc_y[0] << " " << nodeInfoVecs.node_loc_z[0]<< std::endl;
			std::cout<< nodeInfoVecs.node_loc_x[1] << " " << nodeInfoVecs.node_loc_y[1] << " " << nodeInfoVecs.node_loc_z[1]<< std::endl;
			*/
		/*unsigned begin = 479 * generalParams.max_nbr_count;
		unsigned end = begin + generalParams.max_nbr_count;
		for (unsigned i = begin; i < end; i++){
			unsigned id = edgeInfoVecs.global_neighbors[i];
			if (id < generalParams.max_node_count){
				std::cout<<" 479: "<< id <<std::endl;
			}
		}
		begin = 1004 * generalParams.max_nbr_count;
		end = begin + generalParams.max_nbr_count;
		for (unsigned i = begin; i < end; i++){
			unsigned id = edgeInfoVecs.global_neighbors[i];
			if (id < generalParams.max_node_count){
				std::cout<<" 1004: "<< id <<std::endl;
			}
		}*/
	/*	thrust::counting_iterator<unsigned> counterDeLink(0);

		thrust::transform(
						counterDeLink,
						counterDeLink + generalParams.max_node_count,
				nodeInfoVecs.delinksThreadMade.begin(),
			functor_de_link_nodes(
				thrust::raw_pointer_cast(edgeInfoVecs.global_neighbors.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.global_length_zero.data()),
				thrust::raw_pointer_cast(edgeInfoVecs.current_node_edge_count_vec.data()),
				generalParams.max_nbr_count,
				generalParams.max_node_count ) );
*/

		//add links made by above function. links are double counted since threads create links from a->b and b->a
		unsigned num_placed_links = ceil( thrust::reduce(nodeInfoVecs.links_made_individual_thread.begin(),
			nodeInfoVecs.links_made_individual_thread.end(), 0, thrust::plus<unsigned>()) / 2 );

		//std::cout << " num_placed_links: " << num_placed_links << std::endl;


		//sort by increasing. Notice, the sorting must take place for the entire vector since threads write to different places
		thrust::sort_by_key(
			nodeInfoVecs.id_temp_linked_left.begin(),nodeInfoVecs.id_temp_linked_left.end(),
			nodeInfoVecs.id_temp_linked_right.begin(), thrust::greater<unsigned>() );

		thrust::stable_sort_by_key(
			nodeInfoVecs.id_temp_linked_right.begin(),nodeInfoVecs.id_temp_linked_right.end(),
			nodeInfoVecs.id_temp_linked_left.begin(), thrust::greater<unsigned>() );


		//the copy is not needed for the full vector. The only portion copied is 2 * num_placed_links since that many id's were set.
		thrust::copy(nodeInfoVecs.id_temp_linked_right.begin(),
			nodeInfoVecs.id_temp_linked_right.begin() + 2 * num_placed_links,
			nodeInfoVecs.host_id_right.begin() );
		thrust::copy(nodeInfoVecs.id_temp_linked_left.begin(),
			nodeInfoVecs.id_temp_linked_left.begin() + 2 * num_placed_links,
			nodeInfoVecs.host_id_left.begin());



		//old code, keep here in case of issues. use for validation.
		unsigned idL_init = nodeInfoVecs.host_id_left[0];
		unsigned idR_init = nodeInfoVecs.host_id_right[0];


		unsigned count = 0;
		//std::cout << "nodeInfoVecs.id_temp_linked_left.size(): " << nodeInfoVecs.id_temp_linked_left.size() << std::endl;
//		for (unsigned i = 1; i < nodeInfoVecs.id_temp_linked_left.size(); i++) {
//			//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and
//			//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
//			unsigned idL = nodeInfoVecs.host_id_left[i];
//			unsigned idR = nodeInfoVecs.host_id_right[i];
//
//			if ((idL == idL_init) && (idR == idR_init)){
//				count +=1;
//			}
//			else {
//				count = 0;
//			}
//			//reset initial id's
//			idL_init = idL;
//			idR_init = idR;
//
//
//			if ( ((idL != 0) || (idR != 0) ) && (count == 1)) {
//
//				//count edges
//				std::cout<<"placing id: "<< idL<<" " << idR<<std::endl;
//
//
//				nodeInfoVecs.host_edge_left[generalParams.current_edge_count] = (idL);
//				nodeInfoVecs.host_edge_right[generalParams.current_edge_count] = (idR);
//				generalParams.current_edge_count += 1;
//			}
//
//		}
		//end old code

		idL_init = nodeInfoVecs.host_id_left[0];
		idR_init = nodeInfoVecs.host_id_right[0];
		count = 0;
		for (unsigned i = 1; i < 2 * num_placed_links; i++) {
			//add extra edges and preferred lengths. Notice the lower and upper must be added since each imparts force to one single node and
			//not the neighboring node to the edge. This is b/c edges are solved per node and not per edge
			unsigned idL = nodeInfoVecs.host_id_left[i];
			unsigned idR = nodeInfoVecs.host_id_right[i];

			if ((idL == idL_init) && (idR == idR_init)) {
				count += 1;
			}
			else {
				count = 0;
			}
			//reset initial id's
			idL_init = idL;
			idR_init = idR;


			if (((idL != 0) || (idR != 0)) && (count == 1)) {

				//count edges
				//std::cout << "placing id from tester: " << idL << " " << idR << std::endl;
				//std::cout << " total edge count " << generalParams.current_edge_count << std::endl;
				////std::cout<< nodeInfoVecs.node_loc_x[idL] << " " << nodeInfoVecs.node_loc_y[idL] << " " << nodeInfoVecs.node_loc_z[idL]<< std::endl;
				////std::cout<< nodeInfoVecs.node_loc_x[idR] << " " << nodeInfoVecs.node_loc_y[idR] << " " << nodeInfoVecs.node_loc_z[idR]<< std::endl;
				//double dist = sqrt(
				//	(nodeInfoVecs.node_loc_x[idL] - nodeInfoVecs.node_loc_x[idR]) * (nodeInfoVecs.node_loc_x[idL] - nodeInfoVecs.node_loc_x[idR])+
				//	(nodeInfoVecs.node_loc_y[idL] - nodeInfoVecs.node_loc_y[idR]) * (nodeInfoVecs.node_loc_y[idL] - nodeInfoVecs.node_loc_y[idR])+
				//	(nodeInfoVecs.node_loc_z[idL] - nodeInfoVecs.node_loc_z[idR]) * (nodeInfoVecs.node_loc_z[idL] - nodeInfoVecs.node_loc_z[idR]));
				//std::cout<< "distance: " << dist << std::endl;
				nodeInfoVecs.host_edge_left[generalParams.current_edge_count] = (idL);
				nodeInfoVecs.host_edge_right[generalParams.current_edge_count] = (idR);
				generalParams.current_edge_count += 1;
			}

		}

	/*	unsigned globalcount = thrust::count_if(edgeInfoVecs.global_neighbors.begin(), edgeInfoVecs.global_neighbors.end(), is_less_than(generalParams.max_node_count));

		unsigned linksmade = *(thrust::max_element(links_made_individual_thread.begin(), links_made_individual_thread.end() ));
		unsigned delinksmade = *(thrust::max_element(delinksThreadMade.begin(), delinksThreadMade.end() ));
		std::cout<<"max links made this iteration: "<< linksmade << std::endl;
		std::cout<<"max unlinks made this iteration: "<< delinksmade << std::endl;

		std::cout<<"current_edge_count var: "<< generalParams.current_edge_count << std::endl;
		std::cout<<"current_edge_count global "<< globalcount/2 << std::endl;

		unsigned temp= thrust::reduce(	edgeInfoVecs.current_node_edge_count_vec.begin(),
			edgeInfoVecs.current_node_edge_count_vec.end());
		std::cout<<"current_edge_count dev: "<< temp << std::endl;
	*/




};
