#include "hip/hip_runtime.h"
#include <thrust/system_error.h>
#include <thrust/binary_search.h>
#include <thrust/reduce.h>
#include <algorithm>
#include <thrust/replace.h>
#include <thrust/unique.h>
#include <thrust/gather.h>
#include <ostream>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/sort.h>
#include <thrust/transform_reduce.h>
#include <math.h>

#include "storage.h"
#include "system_builder.h"
#include "collagen_elastin_spring.h"
#include "bending_spring.h"
#include "advance_positions.h"
#include "bucket_scheme.h"
#include "link_nodes.h"
#include "external_force.h"
#include "system.h"
#include "functor_misc.h"

using namespace thrust::placeholders;

void System::set_bucket_scheme(){
	init_dim_general(nodeInfoVecs, domainParams, auxVecs, generalParams);
	init_net_inct_bucket(nodeInfoVecs, domainParams, auxVecs, generalParams);
	build_net_inct_bucket(nodeInfoVecs, domainParams, auxVecs, generalParams);
	extend_net_inct_bucket(nodeInfoVecs, domainParams, auxVecs, generalParams);
}

void System::solve_forces() {

	thrust::fill(nodeInfoVecs.node_force_x.begin(),nodeInfoVecs.node_force_x.end(),0);
	thrust::fill(nodeInfoVecs.node_force_y.begin(),nodeInfoVecs.node_force_y.end(),0);
	thrust::fill(nodeInfoVecs.node_force_z.begin(),nodeInfoVecs.node_force_z.end(),0);
	
	if (generalParams.linking == true) {
		link_nodes(nodeInfoVecs, edgeInfoVecs, auxVecs, generalParams);
	}
		
	calc_bending_spring_force(nodeInfoVecs, bendInfoVecs, generalParams);
	  
	calc_spring_force(nodeInfoVecs, edgeInfoVecs, generalParams);
	
	
    //std::cout<<"pre norm: " << std::flush;
	extensionParams.totalAppliedForce = thrust::transform_reduce(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_force_x.begin(),
				nodeInfoVecs.node_force_y.begin(),
				nodeInfoVecs.node_force_z.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_force_x.begin(),
				nodeInfoVecs.node_force_y.begin(),
				nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
			functor_norm(), 0.0, thrust::plus<double>() );
			thrust::transform(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						nodeInfoVecs.node_force_x.begin(),
						nodeInfoVecs.node_force_y.begin(),
						nodeInfoVecs.node_force_z.begin())) + generalParams.max_node_count,
				nodeInfoVecs.sum_forces_on_node.begin(),//save vector
				functor_norm());
	
	extensionParams.applied_force_upper = thrust::transform_reduce(
												thrust::make_zip_iterator(
													thrust::make_tuple(
														nodeInfoVecs.node_upper_selection_pull.begin(),
														nodeInfoVecs.sum_forces_on_node.begin())),
												thrust::make_zip_iterator(
													thrust::make_tuple(
														nodeInfoVecs.node_upper_selection_pull.begin(),
														nodeInfoVecs.sum_forces_on_node.begin())) + generalParams.max_node_count,
												functor_sum_pulled_forces(), 0.0, thrust::plus<double>());
	
	extensionParams.applied_force_lower = thrust::transform_reduce(
												thrust::make_zip_iterator(
													thrust::make_tuple(
														nodeInfoVecs.node_lower_selection_pull.begin(),
														nodeInfoVecs.sum_forces_on_node.begin())),
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_lower_selection_pull.begin(),
															nodeInfoVecs.sum_forces_on_node.begin())) + generalParams.max_node_count,
												functor_sum_pulled_forces(), 0.0, thrust::plus<double>());
};


void System::solve_system() {

	double lastTime = 0.0;
	bool runIters = true;
	std::cout << " setting initial bucket scheme "  << std::endl;
	std::cout<<"magnitudeForce at beginning of simulation: "<< generalParams.magnitudeForce<<std::endl;

	//set initial epsilon
	generalParams.epsilon = (generalParams.epsilon_factor) *
		sqrt(6.0 * edgeInfoVecs.kB * edgeInfoVecs.temperature * generalParams.dt / edgeInfoVecs.viscosity_elastin);
	std::cout<< "Initial Epsilon: " << generalParams.epsilon << std::endl;
	//initialize images
	storage->print_VTK_file();
	storage->save_params();
	
	set_bucket_scheme();
	external_force(
		nodeInfoVecs,
		generalParams,
		extensionParams,
		domainParams);//set initial step and strain parameters
	std::cout<<"starting system" << std::flush;
	while (runIters == true) {

		generalParams.iterationCounter++;
		generalParams.currentTime += generalParams.dt;
		//if (generalParams.iterationCounter % 50 == 0){
			//std::cout << "current iter: " <<generalParams.iterationCounter<<  std::endl;
		set_bucket_scheme();
		//}
		advance_positions(
			nodeInfoVecs,
			generalParams,
			edgeInfoVecs,
      		randVecs);
		
		solve_forces(); //resets and solves forces for next time step
		double maxVel = *(thrust::max_element(nodeInfoVecs.node_vel.begin(), nodeInfoVecs.node_vel.end()));

		thrust::device_vector<double>::iterator iter = thrust::max_element(nodeInfoVecs.node_vel.begin(), nodeInfoVecs.node_vel.end());
		unsigned position = iter - nodeInfoVecs.node_vel.begin();
		double max_val = *iter;
		
		thrust::device_vector<double>::iterator iterx = thrust::max_element(nodeInfoVecs.node_loc_x.begin(), nodeInfoVecs.node_loc_x.end());
		double max_x = *iterx;
		std::cout<<"max x: " << max_x << std::endl;

		if ((generalParams.iterationCounter % 1000) == 0) {
			double currentStrain = (extensionParams.averageUpperStrain - extensionParams.averageLowerStrain) /
			(extensionParams.originAverageUpperStrain - extensionParams.originAverageLowerStrain ) - 1.0;
			if (currentStrain>4.0){
				runIters=false;
			}		
			std::cout<<" current strain: " << currentStrain << std::endl;
			std::cout<<" max velocity: " << maxVel << std::endl;
			std::cout<<" epsilon: " << generalParams.epsilon << std::endl;
			std::cout << " extensionParams.averageUpperStrain: " << extensionParams.averageUpperStrain << std::endl;
			std::cout << " extensionParams.averageLowerStrain: " << extensionParams.averageLowerStrain << std::endl;
			
			std::cout << " extensionParams.applied_force_lower: " << extensionParams.applied_force_lower << std::endl;
			std::cout << " extensionParams.applied_force_upper: " << extensionParams.applied_force_upper << std::endl;
		
		}
		//store sum of all forces on each node. Used in stress calculations

		if ((generalParams.iterationCounter % 5000) == 0) {								
			storage->print_VTK_file();
			storage->save_params();
		}

		if ((maxVel < generalParams.epsilon) && (generalParams.iterationCounter % 50 == 0)) {
			//perform pulling
			external_force(
				nodeInfoVecs,
				generalParams,
				extensionParams,
				domainParams);

			std::cout<<"Maximum vel: "<< maxVel <<std::endl;
			generalParams.magnitudeForce += generalParams.df;
			
		}
		///////////////////////////////////////////////////////////////////////////////
		//EQUILIBRIUM END
		//////////////////////////////////////////////////////////////////////

	}

};

System::System()  {};

void System::assign_storage(std::shared_ptr<Storage> _storage) {
	storage = _storage;
}

void System::initialize_system(HostNodeInfoVecs& hostNodeInfoVecs) {

	std::cout<< "total Edge Count: "<< generalParams.origin_edge_count << std::endl;
	std::cout << "max num nodes: " << generalParams.max_node_count << std::endl;

	nodeInfoVecs.origin_edge_left = hostNodeInfoVecs.host_spring_edge_left;
	nodeInfoVecs.origin_edge_right = hostNodeInfoVecs.host_spring_edge_right;

	set_node_vecs(//calls initDimensionBucketScheme
		hostNodeInfoVecs);

	set_bend_vecs(hostNodeInfoVecs);

	set_edge_vecs(hostNodeInfoVecs);

	set_extras();
};


void System::set_node_vecs(
	HostNodeInfoVecs& hostNodeInfoVecs) {

	randVecs.gaussianData.resize(generalParams.max_node_count);

	nodeInfoVecs.id_edges_made_temp.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);//corresponds to upperAdj vector size plus a single value to hold number of added nodes
	thrust::fill(nodeInfoVecs.id_edges_made_temp.begin(), nodeInfoVecs.id_edges_made_temp.end(), 0);

	nodeInfoVecs.sum_forces_on_node.resize(generalParams.max_node_count);

	nodeInfoVecs.node_upper_selection_pull.resize(generalParams.max_node_count);
	nodeInfoVecs.node_lower_selection_pull.resize(generalParams.max_node_count);

	nodeInfoVecs.node_vel.resize(generalParams.max_node_count);

  	nodeInfoVecs.node_is_collagen.resize(generalParams.max_node_count);
  	nodeInfoVecs.node_is_elastin.resize(generalParams.max_node_count);

	nodeInfoVecs.node_loc_x.resize(generalParams.max_node_count);
	nodeInfoVecs.node_loc_y.resize(generalParams.max_node_count);
	nodeInfoVecs.node_loc_z.resize(generalParams.max_node_count);
	nodeInfoVecs.node_vel_x.resize(generalParams.max_node_count);
	nodeInfoVecs.node_vel_y.resize(generalParams.max_node_count);
	nodeInfoVecs.node_vel_z.resize(generalParams.max_node_count);


	nodeInfoVecs.node_force_x.resize(generalParams.max_node_count);
	nodeInfoVecs.node_force_y.resize(generalParams.max_node_count);
	nodeInfoVecs.node_force_z.resize(generalParams.max_node_count);

	nodeInfoVecs.discretized_edges_strain.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	nodeInfoVecs.discretized_edges_alignment.resize(generalParams.max_node_count * generalParams.max_nbr_count);

	//sized larger for input later
	
	nodeInfoVecs.device_edge_left.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	nodeInfoVecs.device_edge_right.resize(generalParams.max_node_count * generalParams.max_nbr_count);

	nodeInfoVecs.host_edge_left.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	nodeInfoVecs.host_edge_right.resize(generalParams.max_node_count * generalParams.max_nbr_count);


	thrust::fill(nodeInfoVecs.discretized_edges_strain.begin(), nodeInfoVecs.discretized_edges_strain.end(),0.0);
	thrust::fill(nodeInfoVecs.host_edge_right.begin(), nodeInfoVecs.host_edge_right.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.host_edge_left.begin(), nodeInfoVecs.host_edge_left.end(), 0);	//fill force and velocity with zeros for computation.
	thrust::fill(nodeInfoVecs.id_edges_made_temp.begin(), nodeInfoVecs.id_edges_made_temp.end(), 0);

	thrust::fill(nodeInfoVecs.sum_forces_on_node.begin(), nodeInfoVecs.sum_forces_on_node.end(), 0);

	thrust::fill(nodeInfoVecs.node_upper_selection_pull.begin(),
		nodeInfoVecs.node_upper_selection_pull.end(),false);

	thrust::fill(nodeInfoVecs.node_lower_selection_pull.begin(),
		nodeInfoVecs.node_lower_selection_pull.end(),false);

	thrust::copy(hostNodeInfoVecs.host_node_is_collagen.begin(), hostNodeInfoVecs.host_node_is_collagen.end(), nodeInfoVecs.node_is_collagen.begin());
	thrust::copy(hostNodeInfoVecs.host_node_is_elastin.begin(), hostNodeInfoVecs.host_node_is_elastin.end(), nodeInfoVecs.node_is_elastin.begin());
	thrust::copy(hostNodeInfoVecs.host_pos_x.begin(), hostNodeInfoVecs.host_pos_x.end(), nodeInfoVecs.node_loc_x.begin());
	thrust::copy(hostNodeInfoVecs.host_pos_y.begin(), hostNodeInfoVecs.host_pos_y.end(), nodeInfoVecs.node_loc_y.begin());
	thrust::copy(hostNodeInfoVecs.host_pos_z.begin(), hostNodeInfoVecs.host_pos_z.end(), nodeInfoVecs.node_loc_z.begin());

	nodeInfoVecs.links_made_individual_thread.resize(generalParams.max_node_count);

	nodeInfoVecs.id_temp_linked_left.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);
	nodeInfoVecs.id_temp_linked_right.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);
	//copy fixed positions
	nodeInfoVecs.host_id_left.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);
	nodeInfoVecs.host_id_right.resize(generalParams.max_node_count * generalParams.max_links_per_iteration);

	nodeInfoVecs.is_node_fixed.resize(generalParams.max_node_count);
	thrust::fill(nodeInfoVecs.is_node_fixed.begin(), nodeInfoVecs.is_node_fixed.end(), false);

	//now that all the nodes are loaded in, choose the top to apply strain, and fix the bottom

	determine_bounds();

	//at this point all nodes are filled, so we can generate domainParams before seeding dpd particles.
	init_dim_general(
		nodeInfoVecs,
		domainParams,
		auxVecs,
		generalParams);

	//set original parameters for domain. others will be reset as simulation takes place.
	domainParams.origin_min_x = domainParams.min_x;
	domainParams.origin_max_x = domainParams.max_x;
	domainParams.origin_min_y = domainParams.min_y;
	domainParams.origin_max_y = domainParams.max_y;
	domainParams.origin_min_z = domainParams.min_z;
	domainParams.origin_max_z = domainParams.max_z;
	std::cout<< "node count : " <<nodeInfoVecs.node_loc_y.size()<< std::endl;


	auxVecs.id_bucket_net_intc.resize(generalParams.max_node_count);
	auxVecs.id_value_net_intc.resize(generalParams.max_node_count);
	auxVecs.id_value_expanded_net_intc.resize(27 * (generalParams.max_node_count));
	auxVecs.id_bucket_expanded_net_intc.resize(27 *( generalParams.max_node_count));

};

void System::determine_bounds() {
	//determin z positions of nodes to be pulled and fixed.

	thrust::device_vector<double> pos_temp;
	pos_temp.resize(generalParams.max_node_count);
	if (extensionParams.axis == 0) {
		thrust::copy(nodeInfoVecs.node_loc_z.begin(), nodeInfoVecs.node_loc_z.end(), pos_temp.begin());
	}else{
		thrust::copy(nodeInfoVecs.node_loc_x.begin(), nodeInfoVecs.node_loc_x.end(), pos_temp.begin());
	}


	thrust::device_vector<double>::iterator iter_max = thrust::max_element(pos_temp.begin(), pos_temp.end());
	thrust::device_vector<double>::iterator iter_min = thrust::min_element(pos_temp.begin(), pos_temp.end());
	//unsigned position_max = iter - pos_temp.begin();
	double max_val = *iter_max;
	double min_val = *iter_min;
	
	
	double length = max_val - min_val;
	std::cout<<"start end pos_temp: "<< max_val << " "<< min_val <<std::endl;

	//upperLevelAlt pulls 10% default. Set in main.cpp using input
	if (generalParams.pull_percent >= 1.0 || generalParams.pull_percent < 0.0) {
		generalParams.pull_percent=1.0;
		std::cout<<"ERROR PULL PERCENT MUST BE LESS THAN ONE AND LARGER THAN 0.0"<<std::endl;
	}
	double pull_width = generalParams.pull_percent * length;
	double upperLevelAlt = max_val - pull_width;


	double lowerLevel = min_val + pull_width;

	std::cout<<"minimal level final choice for strain choice: " << lowerLevel <<std::endl;

	std::cout<<"maximal level final choice for strain choice: " << upperLevelAlt <<std::endl;

	//apply strain only to original nodes and not added edge subdivision nodes. Set top and bottom

	thrust::replace_if(nodeInfoVecs.node_upper_selection_pull.begin(), nodeInfoVecs.node_upper_selection_pull.begin() + generalParams.origin_node_count,
						pos_temp.begin(),
						IsGreaterThanLevel( upperLevelAlt ), true);

	thrust::replace_if(nodeInfoVecs.node_lower_selection_pull.begin(), nodeInfoVecs.node_lower_selection_pull.begin() + generalParams.origin_node_count,
						pos_temp.begin(),
						IsLessThanLevel( lowerLevel ), true);

	generalParams.numUpperStrainNodes_collagen = thrust::transform_reduce(		
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_upper_selection_pull.begin(),
															nodeInfoVecs.node_is_collagen.begin())),
															
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_upper_selection_pull.begin(),
															nodeInfoVecs.node_is_collagen.begin())) + generalParams.max_node_count,
													IsEqualToOne_and_node_type(),					
													0, thrust::plus<unsigned>());

	generalParams.numUpperStrainNodes_elastin = thrust::transform_reduce(		
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_upper_selection_pull.begin(),
															nodeInfoVecs.node_is_elastin.begin())),
															
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_upper_selection_pull.begin(),
															nodeInfoVecs.node_is_elastin.begin())) + generalParams.max_node_count,
													IsEqualToOne_and_node_type(),					
													0, thrust::plus<unsigned>());								


	generalParams.numLowerStrainNodes_collagen = thrust::transform_reduce(		
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_lower_selection_pull.begin(),
															nodeInfoVecs.node_is_collagen.begin())),
															
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_lower_selection_pull.begin(),
															nodeInfoVecs.node_is_collagen.begin())) + generalParams.max_node_count,
													IsEqualToOne_and_node_type(),					
													0, thrust::plus<unsigned>());

	generalParams.numLowerStrainNodes_elastin = thrust::transform_reduce(		
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_lower_selection_pull.begin(),
															nodeInfoVecs.node_is_elastin.begin())),
															
													thrust::make_zip_iterator(
														thrust::make_tuple(
															nodeInfoVecs.node_lower_selection_pull.begin(),
															nodeInfoVecs.node_is_elastin.begin())) + generalParams.max_node_count,
													IsEqualToOne_and_node_type(),					
													0, thrust::plus<unsigned>());		

	std::cout<<"first try numLowerStrainNodes_elastin: " << generalParams.numLowerStrainNodes_elastin <<std::endl;
	std::cout<<"first try numUpperStrainNodes_elastin: " << generalParams.numUpperStrainNodes_elastin <<std::endl;
	std::cout<<"first try numLowerStrainNodes_collagen: " << generalParams.numLowerStrainNodes_collagen <<std::endl;
	std::cout<<"first try numUpperStrainNodes_collagen: " << generalParams.numUpperStrainNodes_collagen <<std::endl;
													
	//We need to pull the same number of collagen and elastin nodes from the top and bottom. 
	unsigned num_more_collagen = 0;
	if (generalParams.numLowerStrainNodes_collagen > generalParams.numUpperStrainNodes_collagen){
		num_more_collagen = generalParams.numLowerStrainNodes_collagen - generalParams.numUpperStrainNodes_collagen;
	}else { num_more_collagen = generalParams.numUpperStrainNodes_collagen - generalParams.numLowerStrainNodes_collagen; }

	for (unsigned i = 0; i < num_more_collagen; i++){
		for (unsigned j = 0; j < generalParams.max_node_count; j++){
			bool is_collagen = nodeInfoVecs.node_is_collagen[j];
			if (is_collagen){
				if (generalParams.numLowerStrainNodes_collagen > generalParams.numUpperStrainNodes_collagen){		
					bool is_pulled = nodeInfoVecs.node_lower_selection_pull[j];
					if (is_pulled){
						nodeInfoVecs.node_lower_selection_pull[j]=false;
						break;
					}
				}else{
					bool is_pulled = nodeInfoVecs.node_upper_selection_pull[j];
					if (is_pulled){
						nodeInfoVecs.node_upper_selection_pull[j]=false;
						break;
					}
				}
			}
		}
	}
	unsigned num_more_elastin = 0;
	if (generalParams.numLowerStrainNodes_elastin > generalParams.numUpperStrainNodes_elastin){
		num_more_elastin = generalParams.numLowerStrainNodes_elastin - generalParams.numUpperStrainNodes_elastin;
	}else { num_more_elastin = generalParams.numUpperStrainNodes_elastin - generalParams.numLowerStrainNodes_elastin; }
	

	for (unsigned i = 0; i < num_more_elastin; i++){
		for (unsigned j = 0; j < generalParams.max_node_count; j++){
			bool is_elastin = nodeInfoVecs.node_is_elastin[j];
			if (is_elastin){
				if (generalParams.numLowerStrainNodes_elastin > generalParams.numUpperStrainNodes_elastin){		
					bool is_pulled = nodeInfoVecs.node_lower_selection_pull[j];
					if (is_pulled){
						nodeInfoVecs.node_lower_selection_pull[j]=false;
						break;
					}
				}else{
					bool is_pulled = nodeInfoVecs.node_upper_selection_pull[j];
					if (is_pulled){
						nodeInfoVecs.node_upper_selection_pull[j]=false;
						break;
					}
				}
			}
		}
	}
	
	//Test Fix pulled selection and move them 
	/*for (unsigned i = 0; i < nodeInfoVecs.node_lower_selection_pull.size(); i++){
		bool is_upper_pulled = nodeInfoVecs.node_upper_selection_pull[i];
		bool is_lower_pulled = nodeInfoVecs.node_lower_selection_pull[i];
		if (is_upper_pulled || is_lower_pulled){
			nodeInfoVecs.is_node_fixed[i] = true;
			std::cout<<"fixing: " << i << std::endl;
		}
	}*/
	generalParams.numUpperStrainNodes_collagen = thrust::transform_reduce(		
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_upper_selection_pull.begin(),
				nodeInfoVecs.node_is_collagen.begin())),
				
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_upper_selection_pull.begin(),
				nodeInfoVecs.node_is_collagen.begin())) + generalParams.max_node_count,
		IsEqualToOne_and_node_type(),					
		0, thrust::plus<unsigned>());

	generalParams.numUpperStrainNodes_elastin = thrust::transform_reduce(		
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_is_elastin.begin())),
					
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_is_elastin.begin())) + generalParams.max_node_count,
			IsEqualToOne_and_node_type(),					
			0, thrust::plus<unsigned>());								


	generalParams.numLowerStrainNodes_collagen = thrust::transform_reduce(		
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_is_collagen.begin())),
					
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_is_collagen.begin())) + generalParams.max_node_count,
			IsEqualToOne_and_node_type(),					
			0, thrust::plus<unsigned>());

	generalParams.numLowerStrainNodes_elastin = thrust::transform_reduce(		
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_is_elastin.begin())),
					
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_is_elastin.begin())) + generalParams.max_node_count,
			IsEqualToOne_and_node_type(),					
			0, thrust::plus<unsigned>());		
	
	generalParams.numUpperStrainNodes=generalParams.numUpperStrainNodes_elastin + generalParams.numUpperStrainNodes_collagen;
	generalParams.numLowerStrainNodes=generalParams.numLowerStrainNodes_elastin + generalParams.numLowerStrainNodes_collagen;

	std::cout<<"numLowerStrainNodes_elastin: " << generalParams.numLowerStrainNodes_elastin <<std::endl;
	std::cout<<"numUpperStrainNodes_elastin: " << generalParams.numUpperStrainNodes_elastin <<std::endl;
	std::cout<<"numLowerStrainNodes_collagen: " << generalParams.numLowerStrainNodes_collagen <<std::endl;
	std::cout<<"numUpperStrainNodes_collagen: " << generalParams.numUpperStrainNodes_collagen <<std::endl;
	
	unsigned numFixed = thrust::count_if(nodeInfoVecs.is_node_fixed.begin(),nodeInfoVecs.is_node_fixed.end(), IsEqualToOne() );
	std::cout<<"number of nodes fixed: " << numFixed <<std::endl;
	pos_temp.resize(0);

}

void System::set_bend_vecs(
	HostNodeInfoVecs& hostNodeInfoVecs) {

	bendInfoVecs.leftIndex.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.centerIndex.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.rightIndex.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.angleZero.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);

	thrust::fill(bendInfoVecs.leftIndex.begin(),bendInfoVecs.leftIndex.end(),ULONG_MAX);
	thrust::fill(bendInfoVecs.centerIndex.begin(),bendInfoVecs.centerIndex.end(),ULONG_MAX);
	thrust::fill(bendInfoVecs.rightIndex.begin(),bendInfoVecs.rightIndex.end(),ULONG_MAX);

	//after default value is set, set the real id's
	thrust::copy(hostNodeInfoVecs.host_torsion_index_left.begin(), hostNodeInfoVecs.host_torsion_index_left.end(), bendInfoVecs.leftIndex.begin());
	thrust::copy(hostNodeInfoVecs.host_torsion_index_center.begin(), hostNodeInfoVecs.host_torsion_index_center.end(), bendInfoVecs.centerIndex.begin());
	thrust::copy(hostNodeInfoVecs.host_torsion_index_right.begin(), hostNodeInfoVecs.host_torsion_index_right.end(), bendInfoVecs.rightIndex.begin());

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				bendInfoVecs.leftIndex.begin(),
				bendInfoVecs.centerIndex.begin(),
				bendInfoVecs.rightIndex.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				bendInfoVecs.leftIndex.begin(),
				bendInfoVecs.centerIndex.begin(),
				bendInfoVecs.rightIndex.begin())) + bendInfoVecs.total_bend_count,
			bendInfoVecs.angleZero.begin(),//save vector
		functor_initial_angle(
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_y.data()),
			thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data())));

	//		std::cout<<" in NSD device values"<<std::endl;
	/*for (unsigned i = 0; i<bendInfoVecs.total_bend_count; i++) {
		unsigned n0 = bendInfoVecs.leftIndex[i];
		unsigned n1 = bendInfoVecs.centerIndex[i];
		unsigned n2 = bendInfoVecs.rightIndex[i];
		std::cout<< "angle : "<< n0<< " " << n1<< " " << n2<< " " << bendInfoVecs.angleZero[i]<<std::endl;
	}*/

	//3x bigger since each spring affects 3 nodes.
	bendInfoVecs.forceX.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.forceY.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.forceZ.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.tempForceX.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.tempForceY.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.tempForceZ.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);


	thrust::fill(bendInfoVecs.forceX.begin(), bendInfoVecs.forceX.end(), 0.0);
	thrust::fill(bendInfoVecs.forceY.begin(), bendInfoVecs.forceY.end(), 0.0);
	thrust::fill(bendInfoVecs.forceZ.begin(), bendInfoVecs.forceZ.end(), 0.0);

	bendInfoVecs.tempTorIndices.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);
	bendInfoVecs.reducedIds.resize(bendInfoVecs.bend_factor * bendInfoVecs.total_bend_count);


};

void System::set_edge_vecs(
	HostNodeInfoVecs& hostNodeInfoVecs ) {

	edgeInfoVecs.global_neighbors.resize(generalParams.max_node_count * generalParams.max_nbr_count);
  	edgeInfoVecs.global_isedge_collagen.resize(generalParams.max_node_count * generalParams.max_nbr_count);
  	edgeInfoVecs.global_isedge_elastin.resize(generalParams.max_node_count * generalParams.max_nbr_count);

	edgeInfoVecs.current_node_edge_count_vec.resize(generalParams.max_node_count);

	edgeInfoVecs.global_length_zero.resize(generalParams.max_node_count * generalParams.max_nbr_count);
	edgeInfoVecs.num_origin_nbr_per_node_vec.resize(generalParams.max_node_count);


  	thrust::fill(edgeInfoVecs.global_neighbors.begin(), edgeInfoVecs.global_neighbors.end(), generalParams.max_node_count);
  	thrust::fill(edgeInfoVecs.global_isedge_collagen.begin(), edgeInfoVecs.global_isedge_collagen.end(), false);
  	thrust::fill(edgeInfoVecs.global_isedge_elastin.begin(), edgeInfoVecs.global_isedge_elastin.end(), false);

  	thrust::fill(edgeInfoVecs.current_node_edge_count_vec.begin(), edgeInfoVecs.current_node_edge_count_vec.end(),0);
	thrust::fill(edgeInfoVecs.global_length_zero.begin(), edgeInfoVecs.global_length_zero.end(), 0.0);



	nodeInfoVecs.host_edge_left = hostNodeInfoVecs.host_spring_edge_left;
	nodeInfoVecs.host_edge_right = hostNodeInfoVecs.host_spring_edge_right;
	//scan through hostAdj and put in device.
	for (unsigned id = 0; id < hostNodeInfoVecs.host_spring_length_zero.size(); id++) {
		generalParams.totalNumberOfEdges++;
		unsigned idL = hostNodeInfoVecs.host_spring_edge_left[id];
		unsigned idR = hostNodeInfoVecs.host_spring_edge_right[id];

     	bool is_idL_collagen = hostNodeInfoVecs.host_node_is_collagen[idL];
		bool is_idR_collagen = hostNodeInfoVecs.host_node_is_collagen[idR];
		bool is_edge_collagen = false;
		bool is_edge_elastin = false;
		if (is_idL_collagen && is_idR_collagen) {
			is_edge_collagen=true;
		}
		else { is_edge_elastin = true;}
		//std::cout<< "linking " << idL << " to " <<idR << std::endl;

		 double edgeLen = hostNodeInfoVecs.host_spring_length_zero[id];
				//we use the global_length_zero vector to identify edges as well.

    	//node id is row, column node is connected to row node.
		//add edge for left node
		unsigned edgeNumL = edgeInfoVecs.current_node_edge_count_vec[idL]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexL = idL*generalParams.max_nbr_count + edgeNumL;
		edgeInfoVecs.global_length_zero[indexL] = edgeLen;
		edgeInfoVecs.global_neighbors[indexL] = idR;
		edgeInfoVecs.global_isedge_collagen[indexL] = is_edge_collagen;
		edgeInfoVecs.global_isedge_elastin[indexL] = is_edge_elastin;

		(edgeInfoVecs.current_node_edge_count_vec[idL])++; //right connects to left

		//add edge for right node
		unsigned edgeNumR = edgeInfoVecs.current_node_edge_count_vec[idR]; //number of edges on (nodeId = row)	is that entry in cECV
		unsigned indexR = idR*generalParams.max_nbr_count + edgeNumR;
		edgeInfoVecs.global_length_zero[indexR] = edgeLen;
		edgeInfoVecs.global_neighbors[indexR] = idL;
		edgeInfoVecs.global_isedge_collagen[indexR] = is_edge_collagen;
		edgeInfoVecs.global_isedge_elastin[indexR] = is_edge_elastin;

		(edgeInfoVecs.current_node_edge_count_vec[idR])++; //left connects to right
		generalParams.current_edge_count += 1;
	}
	//at this point current_node_edge_count_vec holds the number of edges, copy this to
	thrust::copy(edgeInfoVecs.current_node_edge_count_vec.begin(), edgeInfoVecs.current_node_edge_count_vec.end(), edgeInfoVecs.num_origin_nbr_per_node_vec.begin());
};

void System::set_extras() {
	if (extensionParams.axis==0){
		extensionParams.originalNetworkLength = domainParams.max_z; //compression along x extensionParams.axis
	}else{
		extensionParams.originalNetworkLength = domainParams.max_x; }
};
