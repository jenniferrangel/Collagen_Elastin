#include "system_structures.h"
#include "system.h"
#include "external_force.h"
#include "functor_strain.h"
#include "functor_external_force.h"
#include "functor_external_pull.h"


void external_force(
	NodeInfoVecs& nodeInfoVecs,
	GeneralParams& generalParams,
	ExtensionParams& extensionParams,
	DomainParams& domainParams){

	if ((generalParams.numUpperStrainNodes_collagen > 0) && (generalParams.numLowerStrainNodes_collagen > 0)){
		//try only counting collagen
		extensionParams.averageUpperStrain = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_collagen.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_collagen.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())) + generalParams.max_node_count,
			functor_strain(extensionParams.axis, extensionParams.originalNetworkLength),
				0.0,
			thrust::plus<double>())) / generalParams.numUpperStrainNodes_collagen;
			

		extensionParams.averageLowerStrain = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_collagen.begin(),
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_collagen.begin(),
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())) + generalParams.max_node_count,
			functor_strain(extensionParams.axis, extensionParams.originalNetworkLength),
				0.0,
			thrust::plus<double>())) / generalParams.numLowerStrainNodes_collagen;

			thrust::counting_iterator<unsigned> indexBeginA(0);

			thrust::for_each(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						indexBeginA,
						nodeInfoVecs.node_loc_z.begin(),
						nodeInfoVecs.node_loc_x.begin(),
						nodeInfoVecs.is_node_fixed.begin(),
						nodeInfoVecs.node_is_collagen.begin(),
						nodeInfoVecs.node_upper_selection_pull.begin(),
						nodeInfoVecs.node_lower_selection_pull.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						indexBeginA,
						nodeInfoVecs.node_loc_z.begin(),
						nodeInfoVecs.node_loc_x.begin(),
						nodeInfoVecs.is_node_fixed.begin(),
						nodeInfoVecs.node_is_collagen.begin(),
						nodeInfoVecs.node_upper_selection_pull.begin(),
						nodeInfoVecs.node_lower_selection_pull.begin())) + generalParams.max_node_count,
				functor_external_pull(
					thrust::raw_pointer_cast(nodeInfoVecs.is_node_fixed.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
					generalParams.pull_ammount,
					extensionParams.axis,
					extensionParams.originalNetworkLength,
					extensionParams.strain_proportion_end_sim,
					extensionParams.averageLowerStrain,
					extensionParams.averageUpperStrain));
			
	}
	
	//also pull elastin for now. 
	if ((generalParams.numUpperStrainNodes_elastin >0) && ( generalParams.numLowerStrainNodes_elastin > 0)){
		
		extensionParams.averageUpperStrain = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_elastin.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_elastin.begin(),
					nodeInfoVecs.node_upper_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())) + generalParams.max_node_count,
			functor_strain(extensionParams.axis, extensionParams.originalNetworkLength),
				0.0,
			thrust::plus<double>())) / generalParams.numUpperStrainNodes_elastin;
			
		extensionParams.averageLowerStrain = (thrust::transform_reduce(
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_elastin.begin(),
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())),
			thrust::make_zip_iterator(
				thrust::make_tuple(
					nodeInfoVecs.node_is_elastin.begin(),
					nodeInfoVecs.node_lower_selection_pull.begin(),
					nodeInfoVecs.node_loc_z.begin(),
					nodeInfoVecs.node_loc_x.begin())) + generalParams.max_node_count,
			functor_strain(extensionParams.axis, extensionParams.originalNetworkLength),
				0.0,
			thrust::plus<double>())) / generalParams.numLowerStrainNodes_elastin;
			thrust::counting_iterator<unsigned> indexBeginA(0);

			thrust::for_each(
				thrust::make_zip_iterator(
					thrust::make_tuple(
						indexBeginA,
						nodeInfoVecs.node_loc_z.begin(),
						nodeInfoVecs.node_loc_x.begin(),
						nodeInfoVecs.is_node_fixed.begin(),
						nodeInfoVecs.node_is_elastin.begin(),
						nodeInfoVecs.node_upper_selection_pull.begin(),
						nodeInfoVecs.node_lower_selection_pull.begin())),
				thrust::make_zip_iterator(
					thrust::make_tuple(
						indexBeginA,
						nodeInfoVecs.node_loc_z.begin(),
						nodeInfoVecs.node_loc_x.begin(),
						nodeInfoVecs.is_node_fixed.begin(),
						nodeInfoVecs.node_is_elastin.begin(),
						nodeInfoVecs.node_upper_selection_pull.begin(),
						nodeInfoVecs.node_lower_selection_pull.begin())) + generalParams.max_node_count,
				functor_external_pull(
					thrust::raw_pointer_cast(nodeInfoVecs.is_node_fixed.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.node_loc_x.data()),
					thrust::raw_pointer_cast(nodeInfoVecs.node_loc_z.data()),
					generalParams.pull_ammount,
					extensionParams.axis,
					extensionParams.originalNetworkLength,
					extensionParams.strain_proportion_end_sim,
					extensionParams.averageLowerStrain,
					extensionParams.averageUpperStrain));
	}
	if (generalParams.iterationCounter < 2) {
		extensionParams.originAverageUpperStrain = extensionParams.averageUpperStrain;
		extensionParams.originAverageLowerStrain = extensionParams.averageLowerStrain;
	}

	//Apply External Force. 
	//Currently, we apply forces to all nodes withing a range of the average hight of those chosen for the upper/lower section. 
	//We always apply force to the collagen though. maybe remove that feature? currently collagen is allowed a window of 2micron, elastin 0.5


  };
