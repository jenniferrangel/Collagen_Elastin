#include "hip/hip_runtime.h"
//This file sets the grid for network self interaction

#include "system_structures.h"
#include "bucket_scheme.h"
#include "system.h"

#include "functor_neighbor.h"
#include "functor_bucket_indexer.h"
#include "function_extend.h"

//take domain and discretize into square buckets of size gridspace
void init_dim_general(
	NodeInfoVecs& nodeInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	double minXTemp = (*(thrust::min_element(nodeInfoVecs.node_loc_x.begin(), nodeInfoVecs.node_loc_x.end())));
	double maxXTemp = (*(thrust::max_element(nodeInfoVecs.node_loc_x.begin(), nodeInfoVecs.node_loc_x.end())));
	double minYTemp = (*(thrust::min_element(nodeInfoVecs.node_loc_y.begin(), nodeInfoVecs.node_loc_y.end())));
	double maxYTemp = (*(thrust::max_element(nodeInfoVecs.node_loc_y.begin(), nodeInfoVecs.node_loc_y.end())));
	double minZTemp = (*(thrust::min_element(nodeInfoVecs.node_loc_z.begin(), nodeInfoVecs.node_loc_z.end())));
	double maxZTemp = (*(thrust::max_element(nodeInfoVecs.node_loc_z.begin(), nodeInfoVecs.node_loc_z.end())));


	double space = 0.0;
	domainParams.min_x = minXTemp - space;
	domainParams.max_x = maxXTemp + space;
	domainParams.min_y = minYTemp - space;
	domainParams.max_y = maxYTemp + space;
	domainParams.min_z = minZTemp - space;
	domainParams.max_z = maxZTemp + space;
};

void init_net_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	unsigned padding = 1;
	if (generalParams.iterationCounter == 0) {
		padding = 1;
	}
	else {
		padding = 1;
	}

	//on the first iteration, we allocate more, we don't plan on using it.
	//always set bucket count. Update total if different.
	domainParams.bucket_count_x = padding * ceil((domainParams.max_x - domainParams.min_x) / domainParams.grid_spacing_net_intc) + 1;
	domainParams.bucket_count_y = padding * ceil((domainParams.max_y - domainParams.min_y) / domainParams.grid_spacing_net_intc) + 1;
	domainParams.bucket_count_z = padding * ceil((domainParams.max_z - domainParams.min_z) / domainParams.grid_spacing_net_intc) + 1;

	if ( (domainParams.bucket_count_x * domainParams.bucket_count_x * domainParams.bucket_count_z) > domainParams.total_bucket_count_net_intc	) {
		std::cout<<"resetting grid for network interact" << std::endl;
		std::cout<<"x-bucket: "<< domainParams.bucket_count_x<<std::endl;
		std::cout<<"y-bucket: "<< domainParams.bucket_count_y<<std::endl;
		std::cout<<"z-bucket: "<< domainParams.bucket_count_z<<std::endl;

		//double amount of buckets in case of resizing networks
		domainParams.total_bucket_count_net_intc = 
			domainParams.bucket_count_x * 
			domainParams.bucket_count_y * 
			domainParams.bucket_count_z;
		std::cout<<"grid: "<< domainParams.grid_spacing_net_intc << std::endl;
		std::cout<<"total bucket count: "<< domainParams.total_bucket_count_net_intc<<std::endl;

		std::cout<<"min_x: " << domainParams.min_x << std::endl;
		std::cout<<"max_x: " << domainParams.max_x << std::endl;
		std::cout<<"min_y: " << domainParams.min_y << std::endl;
		std::cout<<"max_y: " << domainParams.max_y << std::endl;
		std::cout<<"min_z: " << domainParams.min_z << std::endl;
		std::cout<<"max_z: " << domainParams.max_z << std::endl;

		auxVecs.key_begin_net_intc.resize(domainParams.total_bucket_count_net_intc);
		auxVecs.key_end_net_intc.resize(domainParams.total_bucket_count_net_intc);

	}

	thrust::fill(auxVecs.key_begin_net_intc.begin(),auxVecs.key_begin_net_intc.end(),0);
	thrust::fill(auxVecs.key_end_net_intc.begin(),auxVecs.key_end_net_intc.end(),0);

};

//convert buckets into neighboring scheme
void extend_net_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {

	//memory is already allocated.
	unsigned endIndexExpanded = (auxVecs.end_index_bucket_keys_net_intc) * 27;


	//test for removing copies.
	unsigned valuesCount = auxVecs.id_value_net_intc.size();
	thrust::fill(auxVecs.id_bucket_expanded_net_intc.begin(),auxVecs.id_bucket_expanded_net_intc.end(),0);
	thrust::fill(auxVecs.id_value_expanded_net_intc.begin(),auxVecs.id_value_expanded_net_intc.end(),0);


	/*
	* beginning of constant iterator
	*/
	thrust::constant_iterator<unsigned> first(27);
	/*
	* end of constant iterator.
	* the plus sign only indicate movement of position, not value.
	* e.g. movement is 5 and first iterator is initialized as 9
	* result array is [9,9,9,9,9];
	*/
	thrust::constant_iterator<unsigned> last = first + (auxVecs.end_index_bucket_keys_net_intc); // this is NOT numerical addition!

	expand(first, last,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_net_intc.begin(),
				auxVecs.id_value_net_intc.begin())),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_net_intc.begin(),
				auxVecs.id_value_expanded_net_intc.begin())));

	thrust::counting_iterator<unsigned> countingBegin(0);

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_net_intc.begin(),
				countingBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_expanded_net_intc.begin(),
				countingBegin)) + endIndexExpanded,

		auxVecs.id_bucket_expanded_net_intc.begin(),
		functor_neighbor(
			domainParams.bucket_count_x,
			domainParams.bucket_count_y,
			domainParams.bucket_count_z));

	thrust::stable_sort_by_key(auxVecs.id_bucket_expanded_net_intc.begin(),
		auxVecs.id_bucket_expanded_net_intc.end(),
		auxVecs.id_value_expanded_net_intc.begin());


	thrust::counting_iterator<unsigned> search_begin(0);
	thrust::counting_iterator<unsigned> search_begin1(0);

	thrust::lower_bound(auxVecs.id_bucket_expanded_net_intc.begin(),
		auxVecs.id_bucket_expanded_net_intc.end(), search_begin,
		search_begin + domainParams.total_bucket_count_net_intc,
		auxVecs.key_begin_net_intc.begin());

	thrust::upper_bound(auxVecs.id_bucket_expanded_net_intc.begin(),
		auxVecs.id_bucket_expanded_net_intc.end(), search_begin1,
		search_begin1 + domainParams.total_bucket_count_net_intc,
		auxVecs.key_end_net_intc.begin());

		
	/*
	unsigned choice = 0;
	unsigned bucket = auxVecs.id_bucket_net_intc[choice];
	std::cout<<"bucketplt 0: "<< bucket<<std::endl;
	std::cout<<"plt pos: "<<nodeInfoVecs.node_loc_x[0]<<" "<<nodeInfoVecs.node_loc_y[0]<<" "<<nodeInfoVecs.node_loc_z[0]<<std::endl;
	std::cout<<"key len: "<< auxVecs.key_begin_net_intc.size() << std::endl;
	unsigned begin = auxVecs.key_begin_net_intc[bucket];
	unsigned end = auxVecs.key_end_net_intc[bucket];
	
	std::cout<<"from bucket scheme:"<<std::endl;
	for (unsigned i = begin; i < end; i++) {
		
		unsigned nbr = auxVecs.id_value_expanded_net_intc[i];
		unsigned buck = auxVecs.id_bucket_net_intc[nbr];
		double x_dist = nodeInfoVecs.node_loc_x[choice] - nodeInfoVecs.node_loc_x[nbr];
		double y_dist = nodeInfoVecs.node_loc_y[choice] - nodeInfoVecs.node_loc_y[nbr];
		double z_dist = nodeInfoVecs.node_loc_z[choice] - nodeInfoVecs.node_loc_z[nbr];
		double dist = std::sqrt(std::pow(x_dist,2.0)+std::pow(y_dist,2.0)+std::pow(z_dist,2.0));
		if ((dist < 19.0) && (dist>0)){
			std::cout<<"dist: "<< dist<< " between: "<< choice << " and nbr: "<< nbr<<std::endl; 
			std::cout<<"nbr: "<< nbr<< " is in bucket: "<< buck <<std::endl;
		}
	}*/

}


//takes nodes and places in buckets.
void build_net_inct_bucket(
	NodeInfoVecs& nodeInfoVecs,
	DomainParams& domainParams,
	AuxVecs& auxVecs,
	GeneralParams& generalParams) {


	thrust::counting_iterator<unsigned> indexBucketBegin(0);
	// takes counting iterator and coordinates
	// return tuple of keys and values
	// transform the points to their bucket indices

	thrust::transform(
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_loc_x.begin(),
				nodeInfoVecs.node_loc_y.begin(),
				nodeInfoVecs.node_loc_z.begin(),
				indexBucketBegin)),
		thrust::make_zip_iterator(
			thrust::make_tuple(
				nodeInfoVecs.node_loc_x.begin(),
				nodeInfoVecs.node_loc_y.begin(),
				nodeInfoVecs.node_loc_z.begin(),
				indexBucketBegin)) + generalParams.max_node_count,
		thrust::make_zip_iterator(
			thrust::make_tuple(
				auxVecs.id_bucket_net_intc.begin(),
				auxVecs.id_value_net_intc.begin())),
		functor_bucket_indexer(
			domainParams.min_x, domainParams.max_x, domainParams.min_y,
			domainParams.max_y, domainParams.min_z, domainParams.max_z,
			domainParams.bucket_count_x,
			domainParams.bucket_count_y,
			domainParams.bucket_count_z,
			domainParams.grid_spacing_net_intc));

//test sorting by node instaed of bucket index
thrust::sort_by_key(auxVecs.id_value_net_intc.begin(),
		auxVecs.id_value_net_intc.begin() + generalParams.max_node_count,
		auxVecs.id_bucket_net_intc.begin());

auxVecs.end_index_bucket_keys_net_intc = generalParams.max_node_count;
};
